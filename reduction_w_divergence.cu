
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCKSIZE 256


__global__ void Reduction(float* IN, float* OUT, int size) {
	int gindex = threadIdx.x + blockIdx.x*blockDim.x;
	int t = threadIdx.x;
	__shared__ float partialSum[BLOCKSIZE];


	partialSum[t] = IN[gindex];
	
	//printf("%f\n", partialSum[t]);
	
	
    for (int stride = 1; stride <= blockDim.x; stride *= 2){
	     __syncthreads();
	     if ((t % (stride * 2) == 0) && ((t+stride) < blockDim.x)){
	     	partialSum[t]+= partialSum[t+stride];
	     	//for the first iteration, partial sums --> t0, t2, t4...
	    }
	}
	if (threadIdx.x == 0){
		OUT[blockIdx.x] = partialSum[0];
		//printf("blockId: %d out: %f\n", blockIdx.x, partialSum[0]);
    }
}

__global__ void single_thread(float* OUT, int numBlocks) {
	for (int i = 1; i < numBlocks; i++){
		//printf("OUT: %f\n", OUT[i]);
		OUT[i] += OUT[i-1];
	}
}



double get_clock(){
	struct timeval tv; int ok;
	ok = gettimeofday(&tv, (void *) 0);
	if (ok<0) { printf("gettimeofday error"); }
	return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


int main(void) {
  int size;
  printf("N: ");
  scanf("%d", &size);

  int numBlocks = ceil(1.0 * size/BLOCKSIZE);
  printf("num blocks %d\n", numBlocks);

  float *in, *out, *IN, *OUT;

  in = (float*)malloc(sizeof(float) * size);
  out = (float*)malloc(sizeof(float) * numBlocks);
  hipMalloc(&IN, sizeof(float)*size);
  hipMalloc(&OUT, sizeof(float)*numBlocks);


  for (int i = 0; i < size; i++) {
         in[i] = i;
  }

  hipMemcpy(IN, in, sizeof(float)*size, hipMemcpyHostToDevice);
  hipMemcpy(OUT, out, sizeof(float)*numBlocks, hipMemcpyHostToDevice);


  double t0 = get_clock();
  Reduction<<<numBlocks, BLOCKSIZE>>>(IN, OUT, size);
  single_thread<<<1, 1>>>(OUT, numBlocks);
  hipMemcpy(out, OUT, sizeof(float)*numBlocks, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  double t1 = get_clock();

  printf("time: %f s \n", (t1-t0));
  printf("sum %f \n", out[numBlocks-1]);
  

  #if 0
  for (int i = 0; i < size; i++){
  	printf("%f\n", out[i]);
  }
  #endif

  hipFree(IN);
  hipFree(OUT);
  free(in);
  free(out);
 
  return 0;
}
